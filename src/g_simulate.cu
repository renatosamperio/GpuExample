#include "hip/hip_runtime.h"

#include "g_simulate.h"

__global__ void vectorAdd(const double *dCounterparty, 
                          const double *dCountry, 
                          const double *dIndustry, 
                          const double *dExposure, 
                          const double *dPdth, 
                          double *dLosses,
                          int numRows) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i < numRows) {
    dLosses[i] = dCounterparty[i] + dCountry[i] + dIndustry[i] + dExposure[i] + dPdth[i] ;
  }
}


int g_simulate(Rcpp::NumericMatrix portfolio,  int n_factor, int n_sim ) {
    
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;
    // Initialising device memory pointers
    double *dCounterparty = NULL;
    double *dCountry      = NULL;
    double *dIndustry     = NULL;
    double *dExposure     = NULL;
    double *dPdth         = NULL;
    double *dLosses       = NULL;

    int numRows = portfolio.nrow();
    int numCols = portfolio.ncol();
    Rcpp::Rcout << "Matrix size ("<< numRows<<", "<<numCols<<"): " 
                << n_factor << " : " << n_sim
                << std::endl;

    // Expand matrix into Rcpp vectors per column
    Rcpp::NumericVector counterparty = portfolio( Rcpp::_ , 0 );
    Rcpp::NumericVector country      = portfolio( Rcpp::_ , 1 );
    Rcpp::NumericVector industry     = portfolio( Rcpp::_ , 2 );
    Rcpp::NumericVector exposure     = portfolio( Rcpp::_ , 3 );
    Rcpp::NumericVector pdTh        = portfolio( Rcpp::_ , 4 );

    // Converting Rcpp to std vector
    std::vector<double> vCounterparty(counterparty.begin(), counterparty.end());
    std::vector<double> vCountry(country.begin(), country.end());
    std::vector<double> vIndustry(industry.begin(), industry.end());
    std::vector<double> vExposure(exposure.begin(), exposure.end());
    std::vector<double> vPdth(pdTh.begin(), pdTh.end());

    // Casting std vector into a pointer
    double* pCounterparty = vCounterparty.data();
    double* pCountry = vCountry.data();
    double* pIndustry = vIndustry.data();
    double* pExposure = vExposure.data();
    double* pPdth = vPdth.data();

    // Copy input data from host to device
    err = hipMalloc((void **)&dCounterparty, numRows);
    err = hipMalloc((void **)&dCountry, numRows);
    err = hipMalloc((void **)&dIndustry, numRows);
    err = hipMalloc((void **)&dExposure, numRows);
    err = hipMalloc((void **)&dPdth, numRows);
    err = hipMalloc((void **)&dLosses, numRows);

    err = hipMemcpy(dCounterparty, pCounterparty, numRows, hipMemcpyHostToDevice);
    err = hipMemcpy(dCountry, pCountry, numRows, hipMemcpyHostToDevice);
    err = hipMemcpy(dIndustry, pIndustry, numRows, hipMemcpyHostToDevice);
    err = hipMemcpy(dExposure, pExposure, numRows, hipMemcpyHostToDevice);
    err = hipMemcpy(dPdth, pPdth, numRows, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numRows + threadsPerBlock - 1) / threadsPerBlock;

    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(dCounterparty, 
                                                  dCountry,
                                                  dIndustry,
                                                  dExposure,
                                                  dPdth,
                                                  dLosses,
                                                  numRows);

    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
                hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return 1;
}