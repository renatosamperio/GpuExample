/*
 * This program uses the host CURAND API to generate 100 
 * pseudorandom floats.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    return EXIT_FAILURE;}} while(0)

int rand_num()
{
    size_t n = 5000000;
    // size_t i;
    hiprandGenerator_t gen;
    float *devData, *hostData;

    /* Allocate n floats on host */
    hostData = (float *)calloc(n, sizeof(float));

    /* Allocate n floats on device */
    CUDA_CALL(hipMalloc((void **)&devData, n*sizeof(float)));

    /* Create pseudo-random number generator */
    CURAND_CALL(hiprandCreateGenerator(&gen, 
                HIPRAND_RNG_PSEUDO_DEFAULT));
    
    /* Set seed */
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
                1ULL));

    /* Generate n floats on device */
    CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(float),
        hipMemcpyDeviceToHost));

    /* Cleanup */
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(devData));
    free(hostData);    
    return EXIT_SUCCESS;
}
